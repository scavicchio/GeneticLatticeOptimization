#include "hip/hip_runtime.h"
//
// Created by Jacob Austin on 5/17/18.
//
#define GLM_FORCE_PURE
#include "spring.h"
#include <cmath>

const double EDGE_DAMPING = 20; // f_damp = delta_v_along_spring*edge_damping_constant;

Vec Spring::getForce() { // computes force on right object. left force is - right force.
  //    Vec temp = (_right -> pos) - (_left -> pos);
  //    return _k * (_rest - temp.norm()) * (temp / temp.norm());

    Vec temp = (_left -> pos) - (_right -> pos);
    Vec spring_force = _k * (temp.norm() - _rest) * (temp / temp.norm());

    //spring_force += dot( (_left->vel - _right->vel) , temp/temp.norm() )*EDGE_DAMPING* (temp/temp.norm());
    return spring_force;
}

int Spring::getLeft() {
    return _left -> index;
}

int Spring::getRight() {
    return _right -> index;
}

void Spring::setForce() { // computes force on right object. left force is - right force.
    Vec f = getForce();
    _right -> force += f;
    _left -> force += -f;
}

// Copy constructor
Spring::Spring(const Spring &other) {
    _k = other._k;
    _rest = other._rest;
    _diam = other._diam;
    _break_force = other._break_force;
    _curr_force = other._curr_force;
    _max_stress = other._max_stress;
    _broken = other._broken;
    _left = nullptr;
    _right = nullptr;
    _actuation = 0.0;
    _compute = true;
}

Spring::Spring(const CUDA_SPRING & spr) {
    this -> _k = spr._k;
    this -> _rest = spr._rest;
    this -> _diam = spr._diam;
    this -> _break_force = spr._break_force;
    this -> _curr_force = spr._curr_force;
    this -> _max_stress = spr._max_stress;
    this -> _broken = spr._broken;
    this -> _actuation = spr._actuation;
    this -> _compute = spr._compute;
}

void Spring::defaultLength() { _rest = (_left -> pos - _right -> pos).norm() ; } //sets Rest Lenght

void Spring::setLeft(Mass * left) {
    if (_left) {
        _left -> decrementRefCount();
    }

    _left = left;
    _left -> ref_count++;

} // sets left mass (attaches spring to mass 1)

void Spring::setRight(Mass * right) {
    if (_right) {
        _right -> decrementRefCount();
    }

    _right = right;
    _right -> ref_count++;
}

void Spring::operator=(CUDA_SPRING & spring) {

    _left = this->_left;
    _right = this->_right;

    _k = spring._k;
    _rest = spring._rest;
    _diam = spring._diam;
    _type = spring._type;
    _period = spring._period;
    _offset = spring._offset;
    _omega = spring._omega;
    _actuation = spring._actuation;
    _max_stress = spring._max_stress;
    _curr_force = spring._curr_force;
    _break_force = spring._break_force;
    _broken = spring._broken;
    _compute = spring._compute;

    arrayptr = this -> arrayptr;
}

CUDA_SPRING::CUDA_SPRING(const Spring & s) {
    _left = (s._left == nullptr) ? nullptr : s._left -> arrayptr;
    _right = (s._right == nullptr) ? nullptr : s. _right -> arrayptr;
    _k = s._k;
    _rest = s._rest;
    _diam = s._diam;
    _type = s._type;
    _period = s._period;
    _offset = s._offset;
    _omega = s._omega;
    _actuation = s._actuation;
    _max_stress = s._max_stress;
    _curr_force = s._curr_force;
    _break_force = s._break_force;
    _broken = s._broken;
    _compute = s._compute;
}

CUDA_SPRING::CUDA_SPRING(const Spring & s, CUDA_MASS * left, CUDA_MASS * right) {
    _left = left;
    _right = right;
    _k = s._k;
    _rest = s._rest;
    _diam = s._diam;
    _type = s._type;
    _period = s._period;
    _offset = s._offset;
    _omega = s._omega;
    _actuation = s._actuation;
    _max_stress = s._max_stress;
    _curr_force = s._curr_force;
    _break_force = s._break_force;
    _broken = s._broken;
    _compute = s._compute;
}
